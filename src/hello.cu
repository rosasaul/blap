

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 512

__global__ void add(int *a, int *b, int *c) {
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

// CPU function to generate a vector of random integers
void random_ints (int *a, int n) {
    for (int i = 0; i < n; i++)
        a[i] = rand() % 10000; // random number between 0 and 9999
}


int main(void) {
  int *a, *b, *c;              // host copies of a, b, c
  int *d_a, *d_b, *d_c;      // device copies of a, b, c
  int size = N * sizeof(int);
              
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);
  
  // Copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  add<<<N,1>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  // Cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  
  return 0;
}

